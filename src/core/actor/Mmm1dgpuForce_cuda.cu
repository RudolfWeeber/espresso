#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2014-2019 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
/** \file
 *  This file contains the code for the polygamma expansions used for the
 *  near formulas of MMM1D on GPU, as well as the force kernels.
 */

#include "config.hpp"

#ifdef MMM1D_GPU

#include "EspressoSystemInterface.hpp"
#include "actor/Mmm1dgpuForce.hpp"
#include "actor/specfunc_cuda.hpp"
#include "cuda_utils.cuh"
#include "electrostatics_magnetostatics/coulomb.hpp"
#include "electrostatics_magnetostatics/mmm-modpsi.hpp"
#include "electrostatics_magnetostatics/mmm1d.hpp"

#include <utils/constants.hpp>

#include <hip/hip_runtime.h>

#include <cstddef>
#include <iostream>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

// the code is mostly multi-GPU capable, but ESPResSo is not yet
const int deviceCount = 1;

#undef hipSetDevice
#define hipSetDevice(d)

__constant__ float far_switch_radius_2[1] = {0.05f * 0.05f};
__constant__ float boxz[1];
__constant__ float uz[1];
__constant__ float coulomb_prefactor[1] = {1.0f};
__constant__ int bessel_cutoff[1] = {5};
__constant__ float maxPWerror[1] = {1e-5f};

// order hardcoded. mmm1d_recalcTables() typically does order less than 30.
// As the coefficients are stored in __constant__ memory, the array needs to be
// sized in advance. We don't know exactly how many coefficients per order, so
// we size plentiful.
const int modpsi_order = 30;
const int modpsi_constant_size = modpsi_order * modpsi_order * 2;

// linearized array on device
__constant__ int device_n_modPsi[1] = {0};
__constant__ int device_linModPsi_offsets[2 * modpsi_order],
    device_linModPsi_lengths[2 * modpsi_order];
__constant__ float device_linModPsi[modpsi_constant_size];

__device__ float dev_mod_psi_even(int n, float x) {
  return evaluateAsTaylorSeriesAt(
      &device_linModPsi[device_linModPsi_offsets[2 * n]],
      device_linModPsi_lengths[2 * n], x * x);
}

__device__ float dev_mod_psi_odd(int n, float x) {
  return x * evaluateAsTaylorSeriesAt(
                 &device_linModPsi[device_linModPsi_offsets[2 * n + 1]],
                 device_linModPsi_lengths[2 * n + 1], x * x);
}

int modpsi_init() {
  create_mod_psi_up_to(modpsi_order);

  // linearized array on host
  std::vector<int> linModPsi_offsets(modPsi.size());
  std::vector<int> linModPsi_lengths(modPsi.size());
  for (size_t i = 0; i < modPsi.size(); i++) {
    if (i)
      linModPsi_offsets[i] =
          linModPsi_offsets[i - 1] + linModPsi_lengths[i - 1];
    linModPsi_lengths[i] = modPsi[i].size();
  }

  // linearize the coefficients array
  std::vector<float> linModPsi(linModPsi_offsets[modPsi.size() - 1] +
                               linModPsi_lengths[modPsi.size() - 1]);
  for (size_t i = 0; i < modPsi.size(); i++) {
    for (size_t j = 0; j < modPsi[i].size(); j++) {
      linModPsi[linModPsi_offsets[i] + j] = static_cast<float>(modPsi[i][j]);
    }
  }

  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);

    // copy to GPU
    int linModPsiSize = linModPsi_offsets[modPsi.size() - 1] +
                        linModPsi_lengths[modPsi.size() - 1];
    if (linModPsiSize > modpsi_constant_size) {
      printf("ERROR: __constant__ device_linModPsi[] is not large enough\n");
      std::abort();
    }
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi_offsets),
                                     linModPsi_offsets.data(),
                                     modPsi.size() * sizeof(int)));
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi_lengths),
                                     linModPsi_lengths.data(),
                                     modPsi.size() * sizeof(int)));
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi), linModPsi.data(),
                                     linModPsiSize * sizeof(float)));
    auto const n_modPsi = static_cast<int>(modPsi.size() >> 1);
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_n_modPsi), &n_modPsi, sizeof(int)));
  }

  return 0;
}

Mmm1dgpuForce::Mmm1dgpuForce(SystemInterface &s, float _coulomb_prefactor,
                             float _maxPWerror, float _far_switch_radius,
                             int _bessel_cutoff)
    : numThreads(64), host_boxz(0), host_npart(0), need_tune(true), pairs(-1),
      dev_forcePairs(nullptr), dev_energyBlocks(nullptr),
      coulomb_prefactor(_coulomb_prefactor), maxPWerror(_maxPWerror),
      far_switch_radius(_far_switch_radius), bessel_cutoff(_bessel_cutoff) {
  // interface sanity checks
  if (!s.requestFGpu())
    throw std::runtime_error("Mmm1dgpuForce needs access to forces on GPU!");

  if (!s.requestRGpu())
    throw std::runtime_error("Mmm1dgpuForce needs access to positions on GPU!");

  if (!s.requestQGpu())
    throw std::runtime_error("Mmm1dgpuForce needs access to charges on GPU!");

  // system sanity checks
  sanity_checks();

  modpsi_init();
}

void Mmm1dgpuForce::setup(SystemInterface &s) {
  if (s.box()[2] <= 0) {
    throw std::runtime_error(
        "Error: Please set box length before initializing MMM1D!");
  }
  if (need_tune && s.npart_gpu() > 0) {
    set_params(static_cast<float>(s.box()[2]),
               static_cast<float>(coulomb.prefactor), maxPWerror,
               far_switch_radius, bessel_cutoff);
    tune(s, maxPWerror, far_switch_radius, bessel_cutoff);
  }
  if (s.box()[2] != host_boxz) {
    set_params(static_cast<float>(s.box()[2]), 0, -1, -1, -1);
  }
  if (s.npart_gpu() == host_npart) // unchanged
  {
    return;
  }

  // For all but the largest systems, it is faster to store force pairs and then
  // sum them up. Atomics are just so slow: so unless we're limited by memory,
  // do the latter.
  pairs = 2;
  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);

    size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    if (freeMem / 2 <
        3 * s.npart_gpu() * s.npart_gpu() *
            sizeof(float)) // don't use more than half the device's memory
    {
      std::cerr << "Switching to atomicAdd due to memory constraints."
                << std::endl;
      pairs = 0;
      break;
    }
  }
  if (dev_forcePairs)
    hipFree(dev_forcePairs);
  if (pairs) // we need memory to store force pairs
  {
    cuda_safe_mem(
        hipMalloc((void **)&dev_forcePairs,
                   3 * s.npart_gpu() * s.npart_gpu() * sizeof(float)));
  }
  if (dev_energyBlocks)
    hipFree(dev_energyBlocks);
  cuda_safe_mem(
      hipMalloc((void **)&dev_energyBlocks, numBlocks(s) * sizeof(float)));
  host_npart = static_cast<int>(s.npart_gpu());
}

unsigned int Mmm1dgpuForce::numBlocks(SystemInterface &s) {
  auto b = static_cast<int>(s.npart_gpu() * s.npart_gpu() / numThreads) + 1;
  if (b > 65535)
    b = 65535;
  return b;
}

Mmm1dgpuForce::~Mmm1dgpuForce() { hipFree(dev_forcePairs); }

__forceinline__ __device__ float sqpow(float x) { return x * x; }
__forceinline__ __device__ float cbpow(float x) { return x * x * x; }

__device__ void sumReduction(float *input, float *sum) {
  auto tid = static_cast<int>(threadIdx.x);
  for (auto i = static_cast<int>(blockDim.x) / 2; i > 0; i /= 2) {
    __syncthreads();
    if (tid < i)
      input[tid] += input[i + tid];
  }
  __syncthreads();
  if (tid == 0)
    sum[0] = input[0];
}

__global__ void sumKernel(float *data, int N) {
  extern __shared__ float partialsums[];
  if (blockIdx.x != 0)
    return;
  auto tid = static_cast<int>(threadIdx.x);
  float result = 0;

  for (int i = 0; i < N; i += static_cast<int>(blockDim.x)) {
    if (i + tid >= N)
      partialsums[tid] = 0;
    else
      partialsums[tid] = data[i + tid];

    sumReduction(partialsums, &result);
    if (tid == 0) {
      if (i == 0)
        data[0] = 0;
      data[0] += result;
    }
  }
}

__global__ void besselTuneKernel(int *result, float far_switch_radius,
                                 int maxCut) {
  const float c_2pif = 2 * Utils::pi<float>();
  float arg = c_2pif * *uz * far_switch_radius;
  float pref = 4 * *uz * max(1.0f, c_2pif * *uz);
  float err;
  int P = 1;
  do {
    err = pref * dev_K1(arg * static_cast<float>(P)) * exp(arg) / arg *
          (static_cast<float>(P) - 1 + 1 / arg);
    P++;
  } while (err > *maxPWerror && P <= maxCut);
  P--;

  result[0] = P;
}

void Mmm1dgpuForce::tune(SystemInterface &s, float _maxPWerror,
                         float _far_switch_radius, int _bessel_cutoff) {
  float far_switch_radius = _far_switch_radius;
  int bessel_cutoff = _bessel_cutoff;
  float maxrad = host_boxz;

  if (_far_switch_radius < 0 && _bessel_cutoff < 0)
  // autodetermine switching radius and Bessel cutoff
  {
    float bestrad = 0, besttime = INFINITY;

    // NOLINTNEXTLINE(clang-analyzer-security.FloatLoopCounter)
    for (far_switch_radius = 0.05f * maxrad; far_switch_radius < maxrad;
         far_switch_radius += 0.05f * maxrad) {
      set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
      tune(s, _maxPWerror, far_switch_radius, -2); // tune Bessel cutoff
      auto runtime = force_benchmark(s);
      if (runtime < besttime) {
        besttime = runtime;
        bestrad = far_switch_radius;
      }
    }
    far_switch_radius = bestrad;

    set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
    tune(s, _maxPWerror, far_switch_radius, -2); // tune Bessel cutoff
  }

  else if (_bessel_cutoff < 0)
  // autodetermine Bessel cutoff
  {
    int *dev_cutoff;
    int maxCut = 30;
    cuda_safe_mem(hipMalloc((void **)&dev_cutoff, sizeof(int)));
    besselTuneKernel<<<dim3(1), dim3(1), 0, nullptr>>>(
        dev_cutoff, far_switch_radius, maxCut);
    cuda_safe_mem(hipMemcpy(&bessel_cutoff, dev_cutoff, sizeof(int),
                             hipMemcpyDeviceToHost));
    hipFree(dev_cutoff);
    if (_bessel_cutoff != -2 &&
        bessel_cutoff >=
            maxCut) // we already have our switching radius and only need to
                    // determine the cutoff, i.e. this is the final tuning round
    {
      throw std::runtime_error(
          "No reasonable Bessel cutoff could be determined.");
    }

    set_params(0, 0, _maxPWerror, far_switch_radius, bessel_cutoff);
  }
}

void Mmm1dgpuForce::set_params(float _boxz, float _coulomb_prefactor,
                               float _maxPWerror, float _far_switch_radius,
                               int _bessel_cutoff, bool manual) {
  if (_boxz > 0 && _far_switch_radius > _boxz) {
    throw std::runtime_error(
        "switching radius must not be larger than box length");
  }
  float _far_switch_radius_2 = _far_switch_radius * _far_switch_radius;
  float _uz = 1.0f / _boxz;
  for (int d = 0; d < deviceCount; d++) {
    // double colons are needed to access the constant memory variables because
    // they are file globals and we have identically named class variables
    hipSetDevice(d);
    if (manual) // tuning needs to be performed again
    {
      far_switch_radius = _far_switch_radius;
      bessel_cutoff = _bessel_cutoff;
    }
    if (_far_switch_radius >= 0) {
      mmm1d_params.far_switch_radius_2 =
          _far_switch_radius * _far_switch_radius;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::far_switch_radius_2),
                                       &_far_switch_radius_2, sizeof(float)));
      far_switch_radius = _far_switch_radius;
    }
    if (_boxz > 0) {
      host_boxz = _boxz;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::boxz), &_boxz, sizeof(float)));
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::uz), &_uz, sizeof(float)));
    }
    if (_coulomb_prefactor != 0) {
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::coulomb_prefactor), &_coulomb_prefactor,
                                       sizeof(float)));
      coulomb_prefactor = _coulomb_prefactor;
    }
    if (_bessel_cutoff > 0) {
      mmm1d_params.bessel_cutoff = _bessel_cutoff;
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::bessel_cutoff), &_bessel_cutoff, sizeof(int)));
      bessel_cutoff = _bessel_cutoff;
    }
    if (_maxPWerror > 0) {
      mmm1d_params.maxPWerror = _maxPWerror;
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::maxPWerror), &_maxPWerror, sizeof(float)));
      maxPWerror = _maxPWerror;
    }
  }
  need_tune = true;

  // The changed parameters in mmm1d_params do not need to be broadcast: they
  // are only accessed by the TCL print function (on node 0) when you call inter
  // coulomb. The CUDA code only runs on node 0, so other nodes do not need the
  // parameters. We couldn't broadcast from here anyway because set_params()
  // might be called from inside computeForces() which is not a time at which
  // the MPI loop on the slave nodes is waiting for broadcasts.
}

__global__ void forcesKernel(const float *__restrict__ r,
                             const float *__restrict__ q,
                             float *__restrict__ force, int N, int pairs,
                             int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  const float c_2pif = 2 * Utils::pi<float>();

  for (int tid =
           static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) + tStart;
       tid < tStop; tid += static_cast<int>(blockDim.x * gridDim.x)) {
    int p1 = tid % N, p2 = tid / N;
    float x = r[3 * p2] - r[3 * p1], y = r[3 * p2 + 1] - r[3 * p1 + 1],
          z = r[3 * p2 + 2] - r[3 * p1 + 2];
    float rxy2 = sqpow(x) + sqpow(y);
    float rxy = sqrt(rxy2);
    float sum_r = 0, sum_z = 0;

    // if (*boxz <= 0.0) return; // in case we are not initialized yet

    while (fabs(z) > *boxz / 2) // make sure we take the shortest distance
      z -= (z > 0 ? 1.f : -1.f) * *boxz;

    if (p1 == p2) // particle exerts no force on itself
    {
      rxy = 1; // so the division at the end doesn't fail with NaN (sum_r is 0
               // anyway)
    } else if (rxy2 <= *far_switch_radius_2) // near formula
    {
      float uzz = *uz * z;
      float uzr = *uz * rxy;
      sum_z = dev_mod_psi_odd(0, uzz);
      float uzrpow = uzr;
      for (int n = 1; n < *device_n_modPsi; n++) {
        float sum_r_old = sum_r;
        float mpe = dev_mod_psi_even(n, uzz);
        float mpo = dev_mod_psi_odd(n, uzz);

        sum_r += 2 * static_cast<float>(n) * mpe * uzrpow;
        uzrpow *= uzr;
        sum_z += mpo * uzrpow;
        uzrpow *= uzr;

        if (fabs(sum_r_old - sum_r) < *maxPWerror)
          break;
      }

      sum_r *= sqpow(*uz);
      sum_z *= sqpow(*uz);

      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      sum_z += z * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_z += (z + *boxz) * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_z += (z - *boxz) * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      if (rxy == 0) // particles at the same radial position only exert a force
                    // in z direction
      {
        rxy = 1; // so the division at the end doesn't fail with NaN (sum_r is 0
                 // anyway)
      }
    } else // far formula
    {
      for (int p = 1; p < *bessel_cutoff; p++) {
        float arg = c_2pif * *uz * static_cast<float>(p);
        sum_r += static_cast<float>(p) * dev_K1(arg * rxy) * cos(arg * z);
        sum_z += static_cast<float>(p) * dev_K0(arg * rxy) * sin(arg * z);
      }
      sum_r *= sqpow(*uz) * 4 * c_2pif;
      sum_z *= sqpow(*uz) * 4 * c_2pif;
      sum_r += 2 * *uz / rxy;
    }

    float pref = *coulomb_prefactor * q[p1] * q[p2];
    if (pairs) {
      force[3 * (p1 + p2 * N - tStart)] = pref * sum_r / rxy * x;
      force[3 * (p1 + p2 * N - tStart) + 1] = pref * sum_r / rxy * y;
      force[3 * (p1 + p2 * N - tStart) + 2] = pref * sum_z;
    } else {
      atomicAdd(&force[3 * p2], pref * sum_r / rxy * x);
      atomicAdd(&force[3 * p2 + 1], pref * sum_r / rxy * y);
      atomicAdd(&force[3 * p2 + 2], pref * sum_z);
    }
  }
}

__global__ void energiesKernel(const float *__restrict__ r,
                               const float *__restrict__ q,
                               float *__restrict__ energy, int N, int pairs,
                               int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  auto const c_2pif = 2 * Utils::pi<float>();
  auto const c_gammaf = Utils::gamma<float>();

  extern __shared__ float partialsums[];
  if (!pairs) {
    partialsums[threadIdx.x] = 0;
    __syncthreads();
  }
  for (int tid =
           static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) + tStart;
       tid < tStop; tid += static_cast<int>(blockDim.x * gridDim.x)) {
    int p1 = tid % N, p2 = tid / N;
    float z = r[3 * p2 + 2] - r[3 * p1 + 2];
    float rxy2 =
        sqpow(r[3 * p2] - r[3 * p1]) + sqpow(r[3 * p2 + 1] - r[3 * p1 + 1]);
    float rxy = sqrt(rxy2);
    float sum_e = 0;

    // if (*boxz <= 0.0) return; // in case we are not initialized yet

    while (fabs(z) > *boxz / 2) // make sure we take the shortest distance
      z -= (z > 0 ? 1.f : -1.f) * *boxz;

    if (p1 == p2) // particle exerts no force on itself
    {
    } else if (rxy2 <= *far_switch_radius_2) // near formula
    {
      float uzz = *uz * z;
      float uzr2 = sqpow(*uz * rxy);
      float uzrpow = uzr2;
      sum_e = dev_mod_psi_even(0, uzz);
      for (int n = 1; n < *device_n_modPsi; n++) {
        float sum_e_old = sum_e;
        float mpe = dev_mod_psi_even(n, uzz);
        sum_e += mpe * uzrpow;
        uzrpow *= uzr2;

        if (fabs(sum_e_old - sum_e) < *maxPWerror)
          break;
      }

      sum_e *= -1 * *uz;
      sum_e -= 2 * *uz * c_gammaf;
      sum_e += rsqrt(rxy2 + sqpow(z));
      sum_e += rsqrt(rxy2 + sqpow(z + *boxz));
      sum_e += rsqrt(rxy2 + sqpow(z - *boxz));
    } else // far formula
    {
      sum_e = -(log(rxy * *uz / 2) + c_gammaf) / 2;
      for (int p = 1; p < *bessel_cutoff; p++) {
        float arg = c_2pif * *uz * static_cast<float>(p);
        sum_e += dev_K0(arg * rxy) * cos(arg * z);
      }
      sum_e *= *uz * 4;
    }

    if (pairs) {
      energy[p1 + p2 * N - tStart] = *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    } else {
      partialsums[threadIdx.x] += *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    }
  }
  if (!pairs) {
    sumReduction(partialsums, &energy[blockIdx.x]);
  }
}

__global__ void vectorReductionKernel(float const *src, float *dst, int N,
                                      int tStart, int tStop) {
  if (tStop < 0)
    tStop = N * N;

  for (auto tid = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
       tid < N; tid += static_cast<int>(blockDim.x * gridDim.x)) {
    int offset = ((tid + (tStart % N)) % N);

    for (int i = 0; tid + i * N < (tStop - tStart); i++) {
#pragma unroll 3
      for (int d = 0; d < 3; d++) {
        dst[3 * offset + d] -= src[3 * (tid + i * N) + d];
      }
    }
  }
}

void Mmm1dgpuForce::computeForces(SystemInterface &s) {
  if (coulomb.method != COULOMB_MMM1D_GPU) {
    std::cerr << "MMM1D: coulomb.method has been changed, skipping calculation"
              << std::endl;
    return;
  }
  setup(s);

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }

  if (pairs) // if we calculate force pairs, we need to reduce them to forces
  {
    auto blocksRed = static_cast<int>(s.npart_gpu() / numThreads) + 1;
    KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
               s.qGpuBegin(), dev_forcePairs, s.npart_gpu(), pairs, 0, -1)
    KERNELCALL(vectorReductionKernel, blocksRed, numThreads, dev_forcePairs,
               s.fGpuBegin(), s.npart_gpu(), 0, -1)
  } else {
    KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
               s.qGpuBegin(), s.fGpuBegin(), s.npart_gpu(), pairs, 0, -1)
  }
}

__global__ void scaleAndAddKernel(float *dst, float const *src, int N,
                                  float factor) {
  for (auto tid = static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x);
       tid < N; tid += static_cast<int>(blockDim.x * gridDim.x)) {
    dst[tid] += src[tid] * factor;
  }
}

void Mmm1dgpuForce::computeEnergy(SystemInterface &s) {
  if (coulomb.method != COULOMB_MMM1D_GPU) {
    std::cerr << "MMM1D: coulomb.method has been changed, skipping calculation"
              << std::endl;
    return;
  }
  setup(s);

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }
  auto shared = static_cast<int>(numThreads * sizeof(float));

  KERNELCALL_shared(energiesKernel, numBlocks(s), numThreads, shared,
                    s.rGpuBegin(), s.qGpuBegin(), dev_energyBlocks,
                    s.npart_gpu(), 0, 0, -1);
  KERNELCALL_shared(sumKernel, 1, numThreads, shared, dev_energyBlocks,
                    numBlocks(s));
  KERNELCALL(scaleAndAddKernel, 1, 1, &(((CUDA_energy *)s.eGpu())->coulomb),
             &dev_energyBlocks[0], 1,
             0.5); // we have counted every interaction twice, so halve the
                   // total energy
}

float Mmm1dgpuForce::force_benchmark(SystemInterface &s) {
  hipEvent_t eventStart, eventStop;
  float elapsedTime;
  float *dev_f_benchmark;

  cuda_safe_mem(
      hipMalloc((void **)&dev_f_benchmark, 3 * s.npart_gpu() * sizeof(float)));
  cuda_safe_mem(hipEventCreate(&eventStart));
  cuda_safe_mem(hipEventCreate(&eventStop));
  cuda_safe_mem(hipEventRecord(eventStart, stream[0]));
  KERNELCALL(forcesKernel, numBlocks(s), numThreads, s.rGpuBegin(),
             s.qGpuBegin(), dev_f_benchmark, s.npart_gpu(), 0, 0, -1)
  cuda_safe_mem(hipEventRecord(eventStop, stream[0]));
  cuda_safe_mem(hipEventSynchronize(eventStop));
  cuda_safe_mem(hipEventElapsedTime(&elapsedTime, eventStart, eventStop));
  cuda_safe_mem(hipEventDestroy(eventStart));
  cuda_safe_mem(hipEventDestroy(eventStop));
  cuda_safe_mem(hipFree(dev_f_benchmark));

  return elapsedTime;
}

#endif /* MMM1D_GPU */
