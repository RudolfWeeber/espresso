#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2014-2022 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */

/** @file
 *  This file contains the code for the polygamma expansions used for the
 *  near formulas of MMM1D on GPU, as well as the force kernels.
 */

#include "config/config.hpp"

#ifdef MMM1D_GPU

#include "electrostatics/mmm-modpsi.hpp"
#include "electrostatics/mmm1d_gpu.hpp"
#include "electrostatics/specfunc.cuh"

#include "EspressoSystemInterface.hpp"
#include "cuda_utils.cuh"

#include <utils/constants.hpp>
#include <utils/math/sqr.hpp>

#include <hip/hip_runtime.h>

#include <cmath>
#include <cstddef>
#include <cstdio>
#include <stdexcept>
#include <vector>

#if defined(OMPI_MPI_H) || defined(_MPI_H)
#error CU-file includes mpi.h! This should not happen!
#endif

// the code is mostly multi-GPU capable, but ESPResSo is not yet
constexpr int deviceCount = 1;

#undef hipSetDevice
#define hipSetDevice(d)

__constant__ float far_switch_radius_sq[1] = {0.05f * 0.05f};
__constant__ float boxz[1];
__constant__ float uz[1];
__constant__ float coulomb_prefactor[1] = {1.0f};
__constant__ int bessel_cutoff[1] = {5};
__constant__ float maxPWerror[1] = {1e-5f};

// As the coefficients are stored in __constant__ memory, the array needs to be
// sized in advance. We don't know exactly how many coefficients per order, so
// we size plentiful.
constexpr int modpsi_order = 30;
constexpr int modpsi_constant_size = modpsi_order * modpsi_order * 2;

// linearized array on device
__constant__ int device_n_modPsi[1] = {0};
__constant__ unsigned int device_linModPsi_offsets[2 * modpsi_order];
__constant__ unsigned int device_linModPsi_lengths[2 * modpsi_order];
__constant__ float device_linModPsi[modpsi_constant_size];

static EspressoSystemInterface *es_system = nullptr;

__device__ float dev_mod_psi_even(int n, float x) {
  return evaluateAsTaylorSeriesAt(
      &device_linModPsi[device_linModPsi_offsets[2 * n]],
      static_cast<int>(device_linModPsi_lengths[2 * n]), x * x);
}

__device__ float dev_mod_psi_odd(int n, float x) {
  return x * evaluateAsTaylorSeriesAt(
                 &device_linModPsi[device_linModPsi_offsets[2 * n + 1]],
                 static_cast<int>(device_linModPsi_lengths[2 * n + 1]), x * x);
}

void CoulombMMM1DGpu::modpsi_init() {
  create_mod_psi_up_to(modpsi_order);

  // linearized array on host
  std::vector<unsigned int> linModPsi_offsets(modPsi.size());
  std::vector<unsigned int> linModPsi_lengths(modPsi.size());
  for (std::size_t i = 0; i < modPsi.size(); i++) {
    if (i)
      linModPsi_offsets[i] =
          linModPsi_offsets[i - 1] + linModPsi_lengths[i - 1];
    linModPsi_lengths[i] = static_cast<unsigned int>(modPsi[i].size());
  }

  // linearize the coefficients array
  std::vector<float> linModPsi(linModPsi_offsets[modPsi.size() - 1] +
                               linModPsi_lengths[modPsi.size() - 1]);
  for (std::size_t i = 0; i < modPsi.size(); i++) {
    for (std::size_t j = 0; j < modPsi[i].size(); j++) {
      linModPsi[linModPsi_offsets[i] + j] = static_cast<float>(modPsi[i][j]);
    }
  }

  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);

    // copy to GPU
    auto const linModPsiSize = linModPsi_offsets[modPsi.size() - 1] +
                               linModPsi_lengths[modPsi.size() - 1];
    if (linModPsiSize > static_cast<unsigned int>(modpsi_constant_size)) {
      throw std::runtime_error(
          "__constant__ device_linModPsi[] is not large enough");
    }
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi_offsets),
                                     linModPsi_offsets.data(),
                                     modPsi.size() * sizeof(int)));
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi_lengths),
                                     linModPsi_lengths.data(),
                                     modPsi.size() * sizeof(int)));
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_linModPsi), linModPsi.data(),
                                     linModPsiSize * sizeof(float)));
    auto const n_modPsi = static_cast<int>(modPsi.size() >> 1);
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(device_n_modPsi), &n_modPsi, sizeof(int)));
  }
}

void CoulombMMM1DGpu::setup() {
  es_system = &EspressoSystemInterface::Instance();
  auto const box_z = static_cast<float>(es_system->box()[2]);
  auto const n_part = es_system->npart_gpu();
  if (not m_is_tuned and n_part != 0) {
    set_params(box_z, prefactor, maxPWerror, far_switch_radius, bessel_cutoff);
    tune(maxPWerror, far_switch_radius, bessel_cutoff);
  }
  if (box_z != host_boxz) {
    set_params(box_z, 0, -1, -1, -1);
  }
  // skip device memory reallocation if device memory is already
  // allocated with the correct vector lengths
  if (n_part == host_npart and pairs != -1) {
    return;
  }
  // For all but the largest systems, it is faster to store force pairs
  // and then sum them up. Atomics are slow, so we only use them when
  // we're limited by device memory, do the latter.
  auto const part_mem_size = 3ul * Utils::sqr(n_part) * sizeof(float);
  pairs = 2;
  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);

    std::size_t freeMem, totalMem;
    hipMemGetInfo(&freeMem, &totalMem);
    if (freeMem / 2 < part_mem_size) {
      // don't use more than half the device's memory
      fprintf(stderr, "Switching to atomicAdd due to memory constraints.\n");
      pairs = 0;
      break;
    }
  }
  if (dev_forcePairs)
    hipFree(dev_forcePairs);
  if (pairs) {
    // we need memory to store force pairs
    cuda_safe_mem(hipMalloc((void **)&dev_forcePairs, part_mem_size));
  }
  if (dev_energyBlocks)
    hipFree(dev_energyBlocks);
  cuda_safe_mem(
      hipMalloc((void **)&dev_energyBlocks, numBlocks() * sizeof(float)));
  host_npart = static_cast<unsigned int>(n_part);
}

unsigned int CoulombMMM1DGpu::numBlocks() const {
  auto b = 1 + static_cast<unsigned int>(Utils::sqr(es_system->npart_gpu()) /
                                         static_cast<std::size_t>(numThreads));
  if (b > 65535)
    b = 65535;
  return b;
}

CoulombMMM1DGpu::~CoulombMMM1DGpu() { hipFree(dev_forcePairs); }

__forceinline__ __device__ float sqpow(float x) { return x * x; }
__forceinline__ __device__ float cbpow(float x) { return x * x * x; }

__device__ void sumReduction(float *input, float *sum) {
  auto const tid = threadIdx.x;
  for (auto i = blockDim.x / 2; i > 0; i /= 2) {
    __syncthreads();
    if (tid < i)
      input[tid] += input[i + tid];
  }
  __syncthreads();
  if (tid == 0)
    sum[0] = input[0];
}

__global__ void sumKernel(float *data, std::size_t N) {
  extern __shared__ float partialsums[];
  if (blockIdx.x != 0)
    return;
  std::size_t const tid = threadIdx.x;
  auto result = 0.f;

  for (std::size_t i = 0; i < N; i += blockDim.x) {
    if (i + tid >= N)
      partialsums[tid] = 0.f;
    else
      partialsums[tid] = data[i + tid];

    sumReduction(partialsums, &result);
    if (tid == 0) {
      if (i == 0)
        data[0] = 0.f;
      data[0] += result;
    }
  }
}

__global__ void besselTuneKernel(int *result, float far_switch_radius,
                                 int maxCut) {
  constexpr auto c_2pif = 2 * Utils::pi<float>();
  auto const arg = c_2pif * *uz * far_switch_radius;
  auto const pref = 4 * *uz * max(1.0f, c_2pif * *uz);
  float err;
  int P = 1;
  do {
    err = pref * dev_K1(arg * static_cast<float>(P)) * exp(arg) / arg *
          (static_cast<float>(P) - 1 + 1 / arg);
    P++;
  } while (err > *maxPWerror && P <= maxCut);
  P--;

  result[0] = P;
}

void CoulombMMM1DGpu::tune(double maxPWerror, double far_switch_radius,
                           int bessel_cutoff) {

  if (far_switch_radius < 0.0 && bessel_cutoff < 0) {
    // autodetermine switching radius and Bessel cutoff
    auto const maxrad = host_boxz;
    auto bestrad = 0.0;
    float besttime = INFINITY;

    // NOLINTNEXTLINE(clang-analyzer-security.FloatLoopCounter)
    for (auto radius = 0.05 * maxrad; radius < maxrad;
         radius += 0.05 * maxrad) {
      set_params(0, 0, maxPWerror, radius, bessel_cutoff);
      tune(maxPWerror, radius, -2); // tune Bessel cutoff
      auto const runtime = force_benchmark();
      if (runtime < besttime) {
        besttime = runtime;
        bestrad = radius;
      }
    }
    set_params(0, 0, maxPWerror, bestrad, bessel_cutoff);
    tune(maxPWerror, bestrad, -2); // tune Bessel cutoff
  } else if (bessel_cutoff < 0) {
    // autodetermine Bessel cutoff
    auto const far_switch_radius_f = static_cast<float>(far_switch_radius);
    int *dev_cutoff;
    constexpr auto maxCut = 30;
    cuda_safe_mem(hipMalloc((void **)&dev_cutoff, sizeof(int)));
    besselTuneKernel<<<dim3(1), dim3(1), 0, nullptr>>>(
        dev_cutoff, far_switch_radius_f, maxCut);
    int best_cutoff = 0;
    cuda_safe_mem(hipMemcpy(&best_cutoff, dev_cutoff, sizeof(int),
                             hipMemcpyDeviceToHost));
    hipFree(dev_cutoff);
    if (bessel_cutoff != -2 && best_cutoff >= maxCut) {
      // we already had our switching radius and only needed to
      // determine the cutoff, i.e. this was the final tuning round
      throw std::runtime_error(
          "No reasonable Bessel cutoff could be determined.");
    }

    set_params(0, 0, maxPWerror, far_switch_radius, best_cutoff);
  }
}

void CoulombMMM1DGpu::set_params(double boxz, double prefactor,
                                 double maxPWerror, double far_switch_radius,
                                 int bessel_cutoff) {
  if (boxz > 0.0 && far_switch_radius > boxz) {
    throw std::runtime_error(
        "switching radius must not be larger than box length");
  }

  for (int d = 0; d < deviceCount; d++) {
    hipSetDevice(d);
    if (far_switch_radius >= 0.0) {
      this->far_switch_radius = far_switch_radius;
      far_switch_radius_sq = Utils::sqr(far_switch_radius);
      auto const far_switch_radius_sq_f =
          static_cast<float>(far_switch_radius_sq);
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::far_switch_radius_sq),
                                       &far_switch_radius_sq_f, sizeof(float)));
    }
    if (boxz > 0.0) {
      host_boxz = static_cast<float>(boxz);
      auto const uz = 1.0f / host_boxz;
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::boxz), &host_boxz, sizeof(float)));
      cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(::uz), &uz, sizeof(float)));
    }
    if (prefactor != 0.0) {
      this->prefactor = prefactor;
      auto const prefactor_f = static_cast<float>(prefactor);
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::coulomb_prefactor), &prefactor_f, sizeof(float)));
    }
    if (bessel_cutoff > 0) {
      this->bessel_cutoff = bessel_cutoff;
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::bessel_cutoff), &bessel_cutoff, sizeof(int)));
    }
    if (maxPWerror > 0.0) {
      this->maxPWerror = maxPWerror;
      auto const maxPWerror_f = static_cast<float>(maxPWerror);
      cuda_safe_mem(
          hipMemcpyToSymbol(HIP_SYMBOL(::maxPWerror), &maxPWerror_f, sizeof(float)));
    }
  }
  m_is_tuned = false;
}

__global__ void forcesKernel(const float *__restrict__ r,
                             const float *__restrict__ q,
                             float *__restrict__ force, std::size_t N,
                             int pairs) {

  constexpr auto c_2pif = 2.f * Utils::pi<float>();
  auto const tStop = Utils::sqr(N);

  for (std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < tStop;
       tid += blockDim.x * gridDim.x) {
    auto const p1 = tid % N, p2 = tid / N;
    auto x = r[3 * p2 + 0] - r[3 * p1 + 0];
    auto y = r[3 * p2 + 1] - r[3 * p1 + 1];
    auto z = r[3 * p2 + 2] - r[3 * p1 + 2];
    auto const rxy2 = sqpow(x) + sqpow(y);
    auto rxy = sqrt(rxy2);
    auto sum_r = 0.f;
    auto sum_z = 0.f;

    while (fabs(z) > *boxz / 2.f) // make sure we take the shortest distance
      z -= (z > 0.f ? 1.f : -1.f) * *boxz;

    if (p1 == p2) {
      // particle exerts no force on itself
      rxy = 1.f; // so the division at the end doesn't fail with NaN
                 // (sum_r is 0 anyway)
    } else if (rxy2 <= *far_switch_radius_sq) {
      // near formula
      auto const uzz = *uz * z;
      auto const uzr = *uz * rxy;
      sum_z = dev_mod_psi_odd(0, uzz);
      auto uzrpow = uzr;
      for (int n = 1; n < *device_n_modPsi; n++) {
        auto const sum_r_old = sum_r;
        auto const mpe = dev_mod_psi_even(n, uzz);
        auto const mpo = dev_mod_psi_odd(n, uzz);

        sum_r += 2 * static_cast<float>(n) * mpe * uzrpow;
        uzrpow *= uzr;
        sum_z += mpo * uzrpow;
        uzrpow *= uzr;

        if (fabs(sum_r_old - sum_r) < *maxPWerror)
          break;
      }

      sum_r *= sqpow(*uz);
      sum_z *= sqpow(*uz);

      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_r += rxy * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      sum_z += z * cbpow(rsqrt(rxy2 + sqpow(z)));
      sum_z += (z + *boxz) * cbpow(rsqrt(rxy2 + sqpow(z + *boxz)));
      sum_z += (z - *boxz) * cbpow(rsqrt(rxy2 + sqpow(z - *boxz)));

      if (rxy == 0.f) {
        // particles at the same radial position only exert a force
        // in z direction
        rxy = 1.f; // so the division at the end doesn't fail with NaN
                   // (sum_r is 0 anyway)
      }
    } else {
      // far formula
      for (int p = 1; p < *bessel_cutoff; p++) {
        float arg = c_2pif * *uz * static_cast<float>(p);
        sum_r += static_cast<float>(p) * dev_K1(arg * rxy) * cos(arg * z);
        sum_z += static_cast<float>(p) * dev_K0(arg * rxy) * sin(arg * z);
      }
      sum_r *= sqpow(*uz) * 4.f * c_2pif;
      sum_z *= sqpow(*uz) * 4.f * c_2pif;
      sum_r += 2.f * *uz / rxy;
    }

    auto const pref = *coulomb_prefactor * q[p1] * q[p2];
    if (pairs) {
      force[3 * (p1 + p2 * N) + 0] = pref * sum_r / rxy * x;
      force[3 * (p1 + p2 * N) + 1] = pref * sum_r / rxy * y;
      force[3 * (p1 + p2 * N) + 2] = pref * sum_z;
    } else {
      atomicAdd(&force[3 * p2 + 0], pref * sum_r / rxy * x);
      atomicAdd(&force[3 * p2 + 1], pref * sum_r / rxy * y);
      atomicAdd(&force[3 * p2 + 2], pref * sum_z);
    }
  }
}

__global__ void energiesKernel(const float *__restrict__ r,
                               const float *__restrict__ q,
                               float *__restrict__ energy, std::size_t N,
                               int pairs) {

  constexpr auto c_2pif = 2.f * Utils::pi<float>();
  constexpr auto c_gammaf = Utils::gamma<float>();
  auto const tStop = Utils::sqr(N);

  extern __shared__ float partialsums[];
  if (!pairs) {
    partialsums[threadIdx.x] = 0;
    __syncthreads();
  }
  for (std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < tStop;
       tid += blockDim.x * gridDim.x) {
    auto const p1 = tid % N, p2 = tid / N;
    auto z = r[3 * p2 + 2] - r[3 * p1 + 2];
    auto const rxy2 = sqpow(r[3 * p2 + 0] - r[3 * p1 + 0]) +
                      sqpow(r[3 * p2 + 1] - r[3 * p1 + 1]);
    auto rxy = sqrt(rxy2);
    auto sum_e = 0.f;

    while (fabs(z) > *boxz / 2.f) // make sure we take the shortest distance
      z -= (z > 0.f ? 1.f : -1.f) * *boxz;

    if (p1 == p2) // particle exerts no force on itself
    {
    } else if (rxy2 <= *far_switch_radius_sq) // near formula
    {
      auto const uzz = *uz * z;
      auto const uzr2 = sqpow(*uz * rxy);
      auto uzrpow = uzr2;
      sum_e = dev_mod_psi_even(0, uzz);
      for (int n = 1; n < *device_n_modPsi; n++) {
        auto const sum_e_old = sum_e;
        auto const mpe = dev_mod_psi_even(n, uzz);
        sum_e += mpe * uzrpow;
        uzrpow *= uzr2;

        if (fabs(sum_e_old - sum_e) < *maxPWerror)
          break;
      }

      sum_e *= -1.f * *uz;
      sum_e -= 2.f * *uz * c_gammaf;
      sum_e += rsqrt(rxy2 + sqpow(z));
      sum_e += rsqrt(rxy2 + sqpow(z + *boxz));
      sum_e += rsqrt(rxy2 + sqpow(z - *boxz));
    } else // far formula
    {
      sum_e = -(log(rxy * *uz / 2.f) + c_gammaf) / 2.f;
      for (int p = 1; p < *bessel_cutoff; p++) {
        auto const arg = c_2pif * *uz * static_cast<float>(p);
        sum_e += dev_K0(arg * rxy) * cos(arg * z);
      }
      sum_e *= *uz * 4.f;
    }

    if (pairs) {
      energy[p1 + p2 * N] = *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    } else {
      partialsums[threadIdx.x] += *coulomb_prefactor * q[p1] * q[p2] * sum_e;
    }
  }
  if (!pairs) {
    sumReduction(partialsums, &energy[blockIdx.x]);
  }
}

__global__ void vectorReductionKernel(float const *src, float *dst,
                                      std::size_t N) {

  auto const tStop = Utils::sqr(N);

  for (std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < N;
       tid += blockDim.x * gridDim.x) {
    auto const offset = tid % N;
    for (std::size_t i = 0; tid + i * N < tStop; i++) {
#pragma unroll 3
      for (std::size_t d = 0; d < 3; d++) {
        dst[3 * offset + d] -= src[3 * (tid + i * N) + d];
      }
    }
  }
}

void CoulombMMM1DGpu::add_long_range_forces() {
  setup();

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }

  if (pairs) {
    // if we calculate force pairs, we need to reduce them to forces
    auto const blocksRed =
        1 + static_cast<unsigned>(es_system->npart_gpu() /
                                  static_cast<std::size_t>(numThreads));
    KERNELCALL(forcesKernel, numBlocks(), numThreads, es_system->rGpuBegin(),
               es_system->qGpuBegin(), dev_forcePairs, es_system->npart_gpu(),
               pairs)
    KERNELCALL(vectorReductionKernel, blocksRed, numThreads, dev_forcePairs,
               es_system->fGpuBegin(), es_system->npart_gpu())
  } else {
    KERNELCALL(forcesKernel, numBlocks(), numThreads, es_system->rGpuBegin(),
               es_system->qGpuBegin(), es_system->fGpuBegin(),
               es_system->npart_gpu(), pairs)
  }
}

__global__ void scaleAndAddKernel(float *dst, float const *src, std::size_t N,
                                  float factor) {
  for (std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x; tid < N;
       tid += blockDim.x * gridDim.x) {
    dst[tid] += src[tid] * factor;
  }
}

void CoulombMMM1DGpu::add_long_range_energy() {
  setup();

  if (pairs < 0) {
    throw std::runtime_error("MMM1D was not initialized correctly");
  }

  auto const shared = numThreads * static_cast<unsigned>(sizeof(float));
  KERNELCALL_shared(energiesKernel, numBlocks(), numThreads, shared,
                    es_system->rGpuBegin(), es_system->qGpuBegin(),
                    dev_energyBlocks, es_system->npart_gpu(), 0);
  KERNELCALL_shared(sumKernel, 1, numThreads, shared, dev_energyBlocks,
                    numBlocks());
  // we count every interaction twice, so halve the total energy
  auto constexpr factor = 0.5f;
  KERNELCALL(scaleAndAddKernel, 1, 1,
             &(reinterpret_cast<CUDA_energy *>(es_system->eGpu())->coulomb),
             &dev_energyBlocks[0], 1, factor);
}

float CoulombMMM1DGpu::force_benchmark() {
  hipEvent_t eventStart, eventStop;
  float elapsedTime;
  float *dev_f_benchmark;

  cuda_safe_mem(hipMalloc((void **)&dev_f_benchmark,
                           3ul * es_system->npart_gpu() * sizeof(float)));
  cuda_safe_mem(hipEventCreate(&eventStart));
  cuda_safe_mem(hipEventCreate(&eventStop));
  cuda_safe_mem(hipEventRecord(eventStart, stream[0]));
  KERNELCALL(forcesKernel, numBlocks(), numThreads, es_system->rGpuBegin(),
             es_system->qGpuBegin(), dev_f_benchmark, es_system->npart_gpu(), 0)
  cuda_safe_mem(hipEventRecord(eventStop, stream[0]));
  cuda_safe_mem(hipEventSynchronize(eventStop));
  cuda_safe_mem(hipEventElapsedTime(&elapsedTime, eventStart, eventStop));
  cuda_safe_mem(hipEventDestroy(eventStart));
  cuda_safe_mem(hipEventDestroy(eventStop));
  cuda_safe_mem(hipFree(dev_f_benchmark));

  return elapsedTime;
}

#endif // MMM1D_GPU
