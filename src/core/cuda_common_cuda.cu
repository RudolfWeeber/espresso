/*
 * Copyright (C) 2010-2019 The ESPResSo project
 *
 * This file is part of ESPResSo.
 *
 * ESPResSo is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * ESPResSo is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 */
#include "config.hpp"

#include "ParticleRange.hpp"
#include "cuda_init.hpp"
#include "cuda_interface.hpp"
#include "cuda_utils.cuh"
#include "errorhandling.hpp"

#include "CudaDeviceAllocator.hpp"
#include "CudaHostAllocator.hpp"

#include <utils/constants.hpp>

#include <thrust/device_vector.h>

#include <hip/hip_runtime.h>

#include <cstddef>
#include <cstdio>

extern int this_node;

template <class T>
using device_vector = thrust::device_vector<T, CudaDeviceAllocator<T>>;

static CUDA_global_part_vars global_part_vars_host = {};

template <class T, class A>
T *raw_data_pointer(thrust::device_vector<T, A> &vec) {
  return thrust::raw_pointer_cast(vec.data());
}

template <class SpanLike> size_t byte_size(SpanLike const &v) {
  return v.size() * sizeof(typename SpanLike::value_type);
}

/** struct for particle force */
static device_vector<float> particle_forces_device;
static device_vector<float> particle_torques_device;

/** struct for particle position and velocity */
static device_vector<CUDA_particle_data> particle_data_device;
/** struct for energies */
static CUDA_energy *energy_device = nullptr;

pinned_vector<CUDA_particle_data> particle_data_host;
pinned_vector<float> particle_forces_host;
CUDA_energy energy_host;

pinned_vector<float> particle_torques_host;

hipStream_t stream[1];

void cuda_check_errors_exit(const dim3 &block, const dim3 &grid,
                            const char *function, const char *file,
                            unsigned int line) {
  hipError_t CU_err = hipGetLastError();
  if (CU_err != hipSuccess) {
    fprintf(stderr,
            "%d: error \"%s\" calling %s with dim %d %d %d, grid %d %d "
            "%d in %s:%u\n",
            this_node, hipGetErrorString(CU_err), function, block.x, block.y,
            block.z, grid.x, grid.y, grid.z, file, line);
    errexit();
  }
}

/**
 * @brief Resize a @ref device_vector.
 *
 * Due to a bug in thrust (https://github.com/thrust/thrust/issues/939),
 * resizing or appending to default constructed containers causes undefined
 * behavior by dereferencing a null-pointer for certain types. This
 * function is used instead of the resize member function to side-step
 * the problem. This is done by replacing the existing vector by a new
 * one constructed with the desired size if resizing from capacity zero.
 * Behaves as-if vec.resize(n) was called.
 *
 * @tparam T Type contained in the vector.
 * @param vec Vector To resize.
 * @param n Desired new size of the element.
 */
template <class T> void resize_or_replace(device_vector<T> &vec, size_t n) {
  if (vec.capacity() == 0) {
    vec = device_vector<T>(n);
  } else {
    vec.resize(n);
  }
}

void resize_buffers(size_t number_of_particles) {
  particle_data_host.resize(number_of_particles);
  resize_or_replace(particle_data_device, number_of_particles);

  particle_forces_host.resize(3 * number_of_particles);
  resize_or_replace(particle_forces_device, 3 * number_of_particles);

#ifdef ROTATION
  particle_torques_host.resize(3 * number_of_particles);
  resize_or_replace(particle_torques_device, 3 * number_of_particles);
#endif
}

/**
 * @brief Setup and call particle reallocation from the host.
 * Note that in addition to calling this function the parameters must be
 * broadcast with either:
 * 1. @ref cuda_bcast_global_part_params() (when just being executed on the
 *    head node) or
 * 2. `MPI_Bcast(gpu_get_global_particle_vars_pointer_host(),
 *    sizeof(CUDA_global_part_vars), MPI_BYTE, 0, comm_cart)` (when executed
 *    on all nodes)
 */
void gpu_init_particle_comm() {
  if (this_node == 0 && global_part_vars_host.communication_enabled == 0) {
    try {
      if (cuda_get_n_gpus() == 0) {
        fprintf(stderr, "ERROR: No GPU was found.\n");
        errexit();
      }
      auto const devID = cuda_get_device();
      auto const compute_capability = cuda_check_gpu_compute_capability(devID);
      auto const communication_test = cuda_test_device_access();
      if (compute_capability != ES_OK or communication_test != ES_OK) {
        fprintf(stderr,
                "ERROR: CUDA device %i is not capable of running ESPResSo.\n",
                devID);
        errexit();
      }
    } catch (cuda_runtime_error const &err) {
      fprintf(stderr, "ERROR: %s\n", err.what());
      errexit();
    }
  }
  global_part_vars_host.communication_enabled = 1;
}

Utils::Span<CUDA_particle_data> gpu_get_particle_pointer() {
  return {raw_data_pointer(particle_data_device), particle_data_device.size()};
}
CUDA_global_part_vars *gpu_get_global_particle_vars_pointer_host() {
  return &global_part_vars_host;
}
float *gpu_get_particle_force_pointer() {
  return raw_data_pointer(particle_forces_device);
}
CUDA_energy *gpu_get_energy_pointer() { return energy_device; }
float *gpu_get_particle_torque_pointer() {
  return raw_data_pointer(particle_torques_device);
}

void copy_part_data_to_gpu(ParticleRange particles) {
  if (global_part_vars_host.communication_enabled == 1) {
    cuda_mpi_get_particles(particles, particle_data_host);

    resize_buffers(particle_data_host.size());

    /* get espressomd particle values */
    if (this_node == 0) {
      hipMemsetAsync(raw_data_pointer(particle_forces_device), 0x0,
                      byte_size(particle_forces_device), stream[0]);
#ifdef ROTATION
      hipMemsetAsync(raw_data_pointer(particle_torques_device), 0x0,
                      byte_size(particle_torques_device), stream[0]);
#endif
      hipMemcpyAsync(raw_data_pointer(particle_data_device),
                      particle_data_host.data(), byte_size(particle_data_host),
                      hipMemcpyHostToDevice, stream[0]);
    }
  }
}

/** setup and call kernel to copy particle forces to host
 */
void copy_forces_from_GPU(ParticleRange &particles) {
  if (global_part_vars_host.communication_enabled == 1) {
    /* Copy result from device memory to host memory*/
    if (this_node == 0 && (not particle_forces_device.empty())) {
      thrust::copy(particle_forces_device.begin(), particle_forces_device.end(),
                   particle_forces_host.begin());
#ifdef ROTATION
      thrust::copy(particle_torques_device.begin(),
                   particle_torques_device.end(),
                   particle_torques_host.begin());
#endif
    }

    cuda_mpi_send_forces(
        particles, {particle_forces_host.data(), particle_forces_host.size()},
        {particle_torques_host.data(), particle_torques_host.size()});
  }
}

void clear_energy_on_GPU() {
  if (!global_part_vars_host.communication_enabled)
    return;
  if (energy_device == nullptr)
    cuda_safe_mem(hipMalloc((void **)&energy_device, sizeof(CUDA_energy)));
  cuda_safe_mem(hipMemset(energy_device, 0, sizeof(CUDA_energy)));
}

CUDA_energy copy_energy_from_GPU() {
  if (!global_part_vars_host.communication_enabled)
    return {};
  cuda_safe_mem(hipMemcpy(&energy_host, energy_device, sizeof(CUDA_energy),
                           hipMemcpyDeviceToHost));
  return energy_host;
}

void cuda_safe_mem_exit(hipError_t CU_err, const char *file,
                        unsigned int line) {
  if (CU_err != hipSuccess) {
    fprintf(stderr, "CUDA Memory error at %s:%u.\n", file, line);
    fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(CU_err));
    if (CU_err == hipErrorInvalidValue)
      fprintf(stderr, "You may have tried to allocate zero memory at %s:%u.\n",
              file, line);
    errexit();
  } else {
    CU_err = hipGetLastError();
    if (CU_err != hipSuccess) {
      fprintf(stderr,
              "Error found during memory operation. Possibly however "
              "from a failed operation before. %s:%u.\n",
              file, line);
      printf("CUDA error: %s\n", hipGetErrorString(CU_err));
      if (CU_err == hipErrorInvalidValue)
        fprintf(stderr,
                "You may have tried to allocate zero memory before %s:%u.\n",
                file, line);
      errexit();
    }
  }
}
